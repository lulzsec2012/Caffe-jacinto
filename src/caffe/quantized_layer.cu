#include "hip/hip_runtime.h"
#include "caffe/quantized_layer.hpp"
#include "caffe/quantized_layer.cuh"
#include "hip/hip_runtime.h" //add by ingenic
namespace caffe {


template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::Quantize_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  if (this->layer_param_.has_quantization_param()) {
    //LOG(INFO) << "Quantizing layer: " << this->layer_param_.name();
    const vector<shared_ptr<Blob > >& blobs = this->blobs();
    const QuantizationParameter& param = this->layer_param_.quantization_param();
    if (param.precision() != QuantizationParameter_Precision_FLOAT) {
      // Trim layer input
      for (int i = 0; i < std::min<int>(param.qparam_in_size(),bottom.size()); ++i) {
        if(param.qparam_in(i).quantize()) {
          this->QuantizeLayerInputs_gpu(bottom[i]->mutable_gpu_data<Ftype>(), i, bottom[i]->count());
        }
      }

      // Trim weights - do it only at the start of quantization
      if(param.qparam_w().quantize() && blobs.size() > 0 && param.quantized_infer_count() == 1100) {
        //this->QuantizeWeights_gpu(blobs[0]->mutable_gpu_data<Ftype>(), blobs[0]->mutable_gpu_connectivity<Ftype>(), blobs[0]->count(), true);//connectivity
	if (this->type() == std::string("Convolution") && this->layer_param_.name() == std::string("res3a_branch2a") && 0) {
           //LOG(INFO) << "hello ingenic!: " << this->layer_param_.name();
           //LOG(INFO) << "hello ingenic!: " << this->layer_param_.name()<< "blobs[0]->is_current_connectivity_valid(): " << blobs[0]->is_current_connectivity_valid();	
           //this->QuantizeWeights_gpu(blobs[0]->mutable_gpu_data<Ftype>(), blobs[0]->mutable_gpu_connectivity<Ftype>(), blobs[0]->count(), true);//connectivity
	}else{
           //LOG(INFO) << "hello world!: " << this->layer_param_.name()<< "blobs[0]->is_current_connectivity_valid(): " << blobs[0]->is_current_connectivity_valid();	
           //this->QuantizeWeights_gpu(blobs[0]->mutable_gpu_data<Ftype>(), blobs[0]->count(), true);
	}
	//this->QuantizeWeights_gpu(blobs[0]->mutable_gpu_data<Ftype>(), blobs[0]->count(), true);
	///this->QuantizeWeights_gpu(blobs[0]->mutable_gpu_data<Ftype>(), blobs[0]->count(), true);
        //if (blobs.size() > 1) { //(this->bias_term_) {
        //  this->QuantizeWeights_gpu(blobs[1]->mutable_gpu_data<Ftype>(), blobs[1]->count(), false);
        //}
      }

      // Trim layer output
      if(param.qparam_out().quantize()) {
        for (int i = 0; i < top.size(); ++i) {
          this->QuantizeLayerOutputs_gpu(top[i]->mutable_gpu_data<Ftype>(), top[i]->count());
        }
      }
    }
  }
}

//add by ingenic
template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::QuantizeWeights_gpu(Ftype* data, Ftype* connectivity, const int count, bool clip) {
  LOG(INFO) << "hello here!: qiangzhileixingzhuangh";     
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  const QuantizationParameter::QParams& qparam_w = param.qparam_w();
  switch (param.precision()) {
  case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
    //Trim2INQ_gpu(data, connectivity, count, qparam_w.bitwidth(), qparam_w.min(), qparam_w.max(), clip);	
    Trim2FixedPoint_gpu(data, count, param.power2_range(), qparam_w.bitwidth(),
        param.rounding_scheme(), qparam_w.fracbits(), qparam_w.scale(),
        qparam_w.offset(), qparam_w.unsigned_quant(), clip);
    break;
  case QuantizationParameter_Precision_FLOAT:
	  break;
  default:
    LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
    break;
  }
}
//~add by ingenic

template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::QuantizeWeights_gpu(Ftype* data, const int count, bool clip) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  const QuantizationParameter::QParams& qparam_w = param.qparam_w();
  switch (param.precision()) {
  case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
    Trim2FixedPoint_gpu(data, count, param.power2_range(), qparam_w.bitwidth(),
        param.rounding_scheme(), qparam_w.fracbits(), qparam_w.scale(),
        qparam_w.offset(), qparam_w.unsigned_quant(), clip);
    break;
  case QuantizationParameter_Precision_FLOAT:
	  break;
  default:
    LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
    break;
  }
}

template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::QuantizeLayerInputs_gpu(
    Ftype* data, const int blob_id, const int count) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  const QuantizationParameter::QParams& qparam_in = param.qparam_in(blob_id);
  switch (param.precision()) {
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, param.power2_range(), qparam_in.bitwidth(),
          param.rounding_scheme(), qparam_in.fracbits(), qparam_in.scale(),
          qparam_in.offset(), qparam_in.unsigned_quant(), true);
      break;
    case QuantizationParameter_Precision_FLOAT:
  	  break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
      break;
  }
}

template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::QuantizeLayerOutputs_gpu(Ftype* data,
      const int count) {
  const QuantizationParameter& param = this->layer_param_.quantization_param();
  const QuantizationParameter::QParams& qparam_out = param.qparam_out();
  switch (param.precision()) {
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, param.power2_range(), qparam_out.bitwidth(),
          param.rounding_scheme(), qparam_out.fracbits(), qparam_out.scale(),
          qparam_out.offset(), qparam_out.unsigned_quant(), true);
      break;
    case QuantizationParameter_Precision_FLOAT:
  	  break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << param.precision() << " for layer:" << this->layer_param_.name();
      break;
  }
}

template <typename Dtype>
__global__ void Trim2FixedPoint_kernel(Dtype* data, const int cnt,
    const int bitwidth, const int rounding, float scale, float inv_scale, float offset, float min_data, float max_data, bool clip) {
    CUDA_KERNEL_LOOP(index, cnt) {

    data[index] = (data[index] * scale) + offset;

    // Round data
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      data[index] = rint(data[index]);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      data[index] = __float2int_rd(data[index] + RandUniform_device(index));
      break;
    default:
      break;
    }

    // Saturate data
    if(clip) {
      data[index] = (data[index]>(Dtype)max_data? (Dtype)max_data:
        (data[index]<(Dtype)min_data?(Dtype)min_data:data[index]));
    }

    data[index] = (data[index] - offset) * inv_scale;
  }
}

template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::Trim2FixedPoint_gpu(Ftype* data, const int cnt, bool power2_range,
      const int bitwidth, const int rounding, int fracbits, float scale, float offset, bool unsigned_quant, bool clip) {
  float inv_scale = 1.0f/scale;

  int qrange = unsigned_quant? bitwidth :  (bitwidth - 1);
  float min_data = unsigned_quant? 0 : -(powf(2, qrange));
  float max_data = +(powf(2, qrange) - 1);

  Trim2FixedPoint_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bitwidth, rounding, scale, inv_scale, offset, min_data, max_data, clip);
}

//add by ingenic
template <typename Dtype>
__global__ void Trim2FixedPoint_kernel_KL(Dtype* data, Dtype* data_tmp, const int cnt,
    const int bitwidth, const int rounding, float scale, float inv_scale, float offset, float min_data, float max_data, bool clip, float * KL_loss) {
    CUDA_KERNEL_LOOP(index, cnt) {

    data_tmp[index] = (data[index] * scale) + offset;

    // Round data
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      data_tmp[index] = rint(data_tmp[index]);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      data_tmp[index] = __float2int_rd(data_tmp[index] + RandUniform_device(index));
      break;
    default:
      break;
    }

    // Saturate data
    if(clip) {
      data_tmp[index] = (data_tmp[index]>(Dtype)max_data? (Dtype)max_data:
        (data_tmp[index]<(Dtype)min_data?(Dtype)min_data:data_tmp[index]));
    }

    data_tmp[index] = (data_tmp[index] - offset) * inv_scale;
    &KL_loss += abs(data_tmp[index] - data[index]);
  }
}

template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::Trim2FixedPoint_gpu_KL(Ftype* data, const int cnt, bool power2_range,
      const int bitwidth, const int rounding, int fracbits, float scale, float offset, bool unsigned_quant, bool clip) {
  float inv_scale = 1.0f/scale;

  int qrange = unsigned_quant? bitwidth :  (bitwidth - 1);
  float min_data = unsigned_quant? 0 : -(powf(2, qrange));
  float max_data = +(powf(2, qrange) - 1);

  //
  int *data_tmp = 0;  
  hipError_t cudaStatus = hipMalloc((void**)&data_tmp, cnt * sizeof(Ftype));  
  if (cudaStatus != hipSuccess) {  
     fprintf(stderr, "hipMalloc failed!");
     LOG(FATAL) << "hipMalloc failed!" << " for layer:" << this->layer_param_.name();
     hipFree(data_tmp);  
    }  
  float min_factor=0.5;
  float max_factor=1.2;
  int   step = 100;
  float factor_step = (max_factor - min_factor)/step;
  float best_KL_loss = 10000;
  int best_step = 0;//71
  for(int i=0;i<step;i++){
     scale_cur = scale * (max_factor + factor_step*i);
     inv_scale = 1.0f/scale_cur;
     float kl_loss = cal_KL_loss();
     if(best_KL_loss > kl_loss){
       best_KL_loss = kl_loss;
       best_step = i;
     }
  }
  if(data_tmp != NULL){
    hipFree(data_tmp);
  }

  LOG(FATAL) << "Best KL factor is:" << max_factor + factor_step*best_step << " for layer:" << this->layer_param_.name();
  scale = scale * (max_factor + factor_step*best_step);
  inv_scale = 1.0f/scale;
  Trim2FixedPoint_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bitwidth, rounding, scale, inv_scale, offset, min_data, max_data, clip);
}

//~add by ingenic

//add by ingenic
template <typename Dtype>
__global__ void weightCluster_zero_kernel(Dtype* data, const int M,
    Dtype* connectivity, bool clip, const int cnt) {
    CUDA_KERNEL_LOOP(index, cnt) {

      Dtype weight;
      // Saturate data
      if(clip) {
        weight = max(min(data[index], pow(2,M)), -pow(2,M));
      }else{
        weight = data[index];
      }     
      double min=100;
      double ind=0;
      double flag=1.0;
      if(connectivity[index]==0){
        if(min>std::abs(weight))
          {
	    min=std::abs(weight);
	    flag=0.0;
          }
    
        for(int i=(M-6);i<=M;i++)
          {
       	    if(min>std::abs(weight-pow(2,i)))
	      {
	        min=std::abs(weight-pow(2,i));
	        ind=i;
	        flag=1.0;
	      }
	    if(min>std::abs(weight+pow(2,i)))
	      {
	        min=std::abs(weight+pow(2,i));
	        ind=i;
	        flag=-1.0;
	      }
          }
        data[index] = flag*pow(2,ind);
      }else{
        data[index] = weight;
      } 
    }
  }
  
template<typename Ftype, typename Btype>
void QuantizedLayer<Ftype, Btype>::Trim2INQ_gpu(Ftype* data, Ftype* connectivity, const int cnt, const int bitwidth, const float min, const float max, bool clip) {
  float max_val_abs = std::max(std::fabs(max), std::fabs(min));
  this->QuantizeWeights_gpu(data, cnt, true);	  
  //caculate the n1
  //int n1=(int)floor(log2(max*4.0/3.0));
  //LOG(INFO) << "weightCluster_zero_kernel: " << " for layer:" << this->layer_param_.name();	
  //weightCluster_zero_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(data,n1,connectivity,clip,cnt);
}

//~add by ingenic

template void QuantizedLayer<double, double>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<double, float>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<double, float16>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);

template void QuantizedLayer<float, double>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<float, float>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<float, float16>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);

template void QuantizedLayer<float16, double>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<float16, float>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);
template void QuantizedLayer<float16, float16>::Quantize_gpu(const vector<Blob*>& bottom,const vector<Blob*>& top);


}  // namespace caffe


